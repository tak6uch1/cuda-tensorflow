
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello(){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    printf("%d Hello CUDA!\n", i);
}

int main() {
    hello<<< 2, 4 >>>();
    hipDeviceSynchronize();
    return 0;
}
